// vAdd.cu
//
// driver and kernel call


#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 32
 
__global__ void vAdd_d (int *a_d, int *b_d, int *c_d, int n)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < n)
		c_d[x] = a_d[x] + b_d[x];
}

extern "C" void gpuAdd (int *a, int *b, int *c, int arraySize)
{
	int *a_d, *b_d, *c_d;

	hipMalloc ((void**) &a_d, sizeof(int) * arraySize);
	hipMalloc ((void**) &b_d, sizeof(int) * arraySize);
	hipMalloc ((void**) &c_d, sizeof(int) * arraySize);
	hipMemcpy (a_d, a, sizeof(int) * arraySize, hipMemcpyHostToDevice);
	hipMemcpy (b_d, b, sizeof(int) * arraySize, hipMemcpyHostToDevice);

	vAdd_d <<< ceil((float) arraySize/THREADS_PER_BLOCK), THREADS_PER_BLOCK >>> (a_d, b_d, c_d, arraySize);
	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf ("CUDA error: %s\n", hipGetErrorString(err));
		
	hipMemcpy (c, c_d, sizeof(int) * arraySize, hipMemcpyDeviceToHost);
	hipFree (a_d);
	hipFree (b_d);
	hipFree (c_d);
}

