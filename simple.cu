/*
 * simple.cu
 * includes setup funtion called from "driver" program
 * also includes kernel function 'cu_fillArray()'
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//#include <string.h>

#define BLOCK_SIZE 32
 
// The __global__ directive identifies this function as a kernel
// Note: all kernels must be declared with return type void 
__global__ void cu_fillArray (int *array_d)
{
    int x;

    // Note: CUDA contains several built-in variables
    // blockIdx.x returns the blockId in the x dimension
    // threadIdx.x returns the threadId in the x dimension
    x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    array_d[x] = x;
}


// This function is called from the host computer.
// It manages memory and calls the function that is executed on the GPU
extern "C" void fillArray (int *array, int arraySize)
{
	//a_d is the GPU counterpart of the array that exists in host memory 
	int *array_d;
	hipError_t result;

	// allocate space in the device 
	result = hipMalloc ((void**) &array_d, sizeof(int) * arraySize);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMalloc failed.");
		exit(1);
	}

	//copy the array from host to array_d in the device 
	result = hipMemcpy (array_d, array, sizeof(int) * arraySize, hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed.");
		exit(1);
	}

	// set execution configuration
	dim3 dimblock (BLOCK_SIZE);
	dim3 dimgrid (arraySize/BLOCK_SIZE);

	// actual computation: Call the kernel
	cu_fillArray <<<dimgrid, dimblock>>> (array_d);

	// transfer results back to host
	result = hipMemcpy (array, array_d, sizeof(int) * arraySize, hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed.");
		exit(1);
	}

	// release the memory on the GPU 
	result = hipFree (array_d);
	if (result != hipSuccess) {
		fprintf(stderr, "hipFree failed.");
		exit(1);
	}
}

