/*
 * blockAndThread.cu
 * includes setup funtion called from "driver" program
 * also includes kernel function 'cu_fillArray()'
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 8
 
// The __global__ directive identifies this function as a kernel
// Note: all kernels must be declared with return type void 
__global__ void cu_fillArray (int *block_d, int *thread_d)
{
    int x;

    // Note: CUDA contains several built-in variables
    // blockIdx.x returns the blockId in the x dimension
    // threadIdx.x returns the threadId in the x dimension
    x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	block_d[x] = blockIdx.x;
	thread_d[x] = threadIdx.x;
}


// This function is called from the host computer.
// It manages memory and calls the function that is executed on the GPU
extern "C" void fillArray (int *block, int *thread, int arraySize)
{
	// block_d and thread_d are the GPU counterparts of the arrays that exists in host memory 
	int *block_d;
	int *thread_d;
	hipError_t result;

	// allocate space in the device 
	result = hipMalloc ((void**) &block_d, sizeof(int) * arraySize);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMalloc (block) failed.");
		exit(1);
	}
	result = hipMalloc ((void**) &thread_d, sizeof(int) * arraySize);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMalloc (thread) failed.");
		exit(1);
	}
	
	//copy the arrays from host to the device 
	result = hipMemcpy (block_d, block, sizeof(int) * arraySize, hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host->dev (block) failed.");
		exit(1);
	}
	result = hipMemcpy (thread_d, thread, sizeof(int) * arraySize, hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host->dev (thread) failed.");
		exit(1);
	}
	
	// set execution configuration
	dim3 dimblock (BLOCK_SIZE);
	dim3 dimgrid (arraySize/BLOCK_SIZE);

	// actual computation: Call the kernel
	cu_fillArray <<<dimgrid, dimblock>>> (block_d, thread_d);

	// transfer results back to host
	result = hipMemcpy (block, block_d, sizeof(int) * arraySize, hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host <- dev (block) failed.");
		exit(1);
	}
	result = hipMemcpy (thread, thread_d, sizeof(int) * arraySize, hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host <- dev (thread) failed.");
		exit(1);
	}
	
	// release the memory on the GPU 
	result = hipFree (block_d);
	if (result != hipSuccess) {
		fprintf(stderr, "hipFree (block) failed.");
		exit(1);
	}
	result = hipFree (thread_d);
	if (result != hipSuccess) {
		fprintf(stderr, "hipFree (thread) failed.");
		exit(1);
	}
}

